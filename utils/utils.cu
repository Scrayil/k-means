#include "hip/hip_runtime.h"
// Copyright (c) 2023. Created by Mattia Bennati, a.k.a Scrayil. All rights reserved.

#include <iostream>
#include "utils.cuh"

// FUNCTIONS

int perform_gpu_check() {
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        std::cout << "No GPU detected!" << std::endl;
        return -1;
    }
    else
        return 0;
}

int* get_iteration_threads_and_blocks(int device_index, int num_data_points, int data_points_batch_size) {
    // Gets the total number of THREADS available on the gpu
    hipDeviceProp_t deviceProp{};
    hipGetDeviceProperties(&deviceProp, device_index);
    int threadsPerSM = deviceProp.maxThreadsPerMultiProcessor;
    int SMCount = deviceProp.multiProcessorCount;
    int TOTAL_THREADS = threadsPerSM * SMCount;

    // Number of THREADS per block
    int THREADS = 256;
    int block_data_size = num_data_points + THREADS - 1;
    int cluster_iterations = 1;
    // Used to handle any number of data_points dynamically
    if(data_points_batch_size > 0 && num_data_points > data_points_batch_size || num_data_points > TOTAL_THREADS) {
        if(data_points_batch_size > TOTAL_THREADS || data_points_batch_size <= 0 && num_data_points > TOTAL_THREADS)
            data_points_batch_size = ((TOTAL_THREADS / THREADS) - 1) * THREADS;

        block_data_size = data_points_batch_size + THREADS - 1;
        cluster_iterations = std::ceil(static_cast<double>(num_data_points) / static_cast<double>(data_points_batch_size));
    }

    // Every element remaining after dividing is allocated to an additional block
    int BLOCKS = block_data_size / THREADS;

    return new int[5]{THREADS, BLOCKS, cluster_iterations, TOTAL_THREADS, data_points_batch_size};
}